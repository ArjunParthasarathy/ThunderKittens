#include "hip/hip_runtime.h"
#include "complex_mma.cuh"

#ifdef TEST_WARP_REGISTER_TILE_MMA_COMPLEX

// Need the wrapper so we can do the implicit const conversion for the inputs
template<typename Ker, typename T, int H, int W, int NW, typename... args>
static __global__ void global_cmplx_wrapper_2d(const T *re_input, const T *im_input, T *re_output, T *im_output) {
    Ker::template device_func<H, W, NW, args...>(re_input, im_input, re_output, im_output);
}

struct test_cmplx_mma_AB {
    template<int H, int W, int NW, typename K> using valid = std::bool_constant<NW == 1 && (2*W*H+W*K::value+H*K::value)<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_cmplx_mma_AB";
    template<int H, int W, int NW, typename _K> __host__ static void host_func(
        const std::vector<float> &re_i_ref, const std::vector<float> &im_i_ref,
        std::vector<float> &re_o_ref, std::vector<float> &im_o_ref) {
        constexpr int K = _K::value;

        // ac
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += re_i_ref[i*16*K + k]*re_i_ref[(256*H*K) + k*16*W + j];
                }
                re_o_ref[i*16*W + j] = sum;
            }
        }

        // bd
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += im_i_ref[i*16*K + k]*im_i_ref[(256*H*K) + k*16*W + j];
                }
                // (ac-bd)
                re_o_ref[i*16*W + j] -= sum;
            }
        }
        
        // ad
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += re_i_ref[i*16*K + k]*im_i_ref[(256*H*K) + k*16*W + j];
                }
                im_o_ref[i*16*W + j] = sum;
            }
        }

        // bc
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += im_i_ref[i*16*K + k]*re_i_ref[(256*H*K) + k*16*W + j];
                }
                // (ad + bc)i
                im_o_ref[i*16*W + j] += sum;
            }
        }

    }
    template<int H, int W, int NW, typename _K> __device__ static void device_func(const kittens::bf16 *re_input, const kittens::bf16 *im_input, 
                                                                                    kittens::bf16 *re_output, kittens::bf16 *im_output) {
        constexpr int K = _K::value;
        kittens::rt_cmplx_bf<H, K> a;
        kittens::rt_cmplx_bf<K, W, kittens::ducks::rt_layout::col> b;
        kittens::rt_cmplx_fl<H, W> c;
        kittens::load(a, re_input, im_input, K*16, K*16);
        kittens::load(b, re_input+a.real.num_elements, im_input+a.imag.num_elements, W*16, W*16);
        kittens::zero(c);
        kittens::mma_AB(c, a, b, c);
        kittens::store(re_output, im_output, c, W*16, W*16);
    }
};

// Due to the strange sizes instantiated, we need a custom base wrapper here
template<typename test, int H, int W, int NUM_WORKERS, typename _K, typename... args>
struct cmplx_mma_wrapper_2d {
    static void run(test_data& results) {
        using namespace kittens;
        constexpr int K = _K::value;
        test_info this_result;
        this_result.label = generate_test_name<H,W,NUM_WORKERS,_K,args...>(test::test_identifier);
        if constexpr (test::template valid<H, W, NUM_WORKERS, _K, args...>::value) {
            // initialize
            kittens::bf16 *d_re_i, *d_im_i;
            kittens::bf16 *d_re_o, *d_im_o;
            std::vector<float> re_i_ref((H+W)*K*256);
            std::vector<float> im_i_ref((H+W)*K*256);
            std::vector<float> re_o_ref(H*W*256);
            std::vector<float> im_o_ref(H*W*256);
            initialize(&d_re_i, &d_re_o, re_i_ref, re_o_ref);
            initialize(&d_im_i, &d_im_o, im_i_ref, im_o_ref);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                global_cmplx_wrapper_2d<test), kittens::bf16, H, W, NUM_WORKERS, _K, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            // Can't use global_wrapper_2d b/c it only accepts 2 params and we need 4 for complex-valued function
            global_cmplx_wrapper_2d<test, kittens::bf16, H, W, NUM_WORKERS, _K, args...><<<1, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(d_re_i, d_im_i, d_re_o, d_im_o);
            // fill in correct results on cpu
            test::template host_func<H, W, NUM_WORKERS, _K, args...>(re_i_ref, im_i_ref, re_o_ref, im_o_ref);
            // check and cleanup
            test_result re_result = validate(d_re_i, d_re_o, re_i_ref, re_o_ref, this_result.label + "_real", W*16, 0.02); // mma's sometimes produce small errors. this appears to be hardware.
            test_result im_result = validate(d_im_i, d_im_o, im_i_ref, im_o_ref, this_result.label + "_imag", W*16, 0.02);
            if (re_result == test_result::PASSED && im_result == test_result::PASSED) {
                // TODO change back
                this_result.result = test_result::PASSED;
            } else {
                this_result.result = test_result::FAILED;
            }
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args> using cmplx_mma_sweep_size = loop_h<cmplx_mma_wrapper_2d, test, MAX_H, MAX_W, NUM_WORKERS, MAX_H, args...>;
template<typename test, int MAX_H=8, int MAX_W=8, typename... args> using cmplx_mma_sweep_size_warp = cmplx_mma_sweep_size<test, MAX_H, MAX_W, 1, args...>;


void warp::reg::tile::mma::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/register/tile/mma tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
    cmplx_mma_sweep_size_warp<test_cmplx_mma_AB, SIZE, SIZE, std::integral_constant<int, 1>>::run(results);
    cmplx_mma_sweep_size_warp<test_cmplx_mma_AB, SIZE, SIZE, std::integral_constant<int, 2>>::run(results);
    cmplx_mma_sweep_size_warp<test_cmplx_mma_AB, SIZE, SIZE, std::integral_constant<int, 3>>::run(results);
    cmplx_mma_sweep_size_warp<test_cmplx_mma_AB, SIZE, SIZE, std::integral_constant<int, 4>>::run(results);
}

#endif