#include "hip/hip_runtime.h"
#include "complex_mul.cuh"

#ifdef TEST_WARP_REGISTER_TILE_MUL_COMPLEX

// Need the wrapper so we can do the implicit const conversion for the inputs
template<typename Ker, typename T, int H, int W, int NW, typename... args>
static __global__ void global_cmplx_wrapper_2d(
    const T *re_input, const T *im_input, 
    T *re_output, T *im_output
) {
    Ker::template device_func<H, W, NW, args...>(
        re_input, im_input, 
        re_output, im_output
    );
}

struct test_cmplx_mul {
    // element wise mult
    // template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && W*H<=64>; // this is warp-level
    template<int H, int W, int NW, typename K> using valid = std::bool_constant<NW == 1 && (2*W*H+W*K::value+H*K::value)<=64>; // this is warp-level

    static inline const std::string test_identifier = "reg_complex_mul";
    template<int H, int W, int NW, typename _K> __host__ static void host_func(
        const std::vector<float> &re_i_ref, 
        const std::vector<float> &im_i_ref,
        std::vector<float> &re_o_ref, std::vector<float> &im_o_ref
    ) {
        constexpr int K = _K::value; //unused
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                // real parts for each element
                re_o_ref[i*16*W + j] += (
                    re_i_ref[i*16*W + j]*re_i_ref[i*16*W + j] - 
                    im_i_ref[i*16*W + j]*im_i_ref[i*16*W + j]
                );

                // imaginary parts for each element
                im_o_ref[i*16*W + j] += (
                    re_i_ref[i*16*W + j]*im_i_ref[i*16*W + j] + 
                    im_i_ref[i*16*W + j]*re_i_ref[i*16*W + j]
                );
            }
        }
    }
    template<int H, int W, int NW, typename _K> __device__ static void device_func(
        const kittens::bf16 *input_re, const kittens::bf16 *input_im,
        kittens::bf16 *output_re, kittens::bf16 *output_im
    ) {
        constexpr int K = _K::value;
        kittens::rt_cmplx_bf<H, W> a;
        kittens::rt_cmplx_bf<H, W> c;
        kittens::load(a, input_re, input_im, W*16, W*16);
        kittens::mul(c, a, a);
        kittens::store(output_re, output_im, c, W*16, W*16);
    }
};

// Due to the strange sizes instantiated, we need a custom base wrapper here
template<typename test, int H, int W, int NUM_WORKERS, typename _K, typename... args>
struct cmplx_mul_wrapper_2d {
    static void run(test_data& results) {
        printf("Running cmplx_mul_wrapper_2d\n"); 
        using namespace kittens;
        constexpr int K = _K::value;
        test_info this_result;
        this_result.label = generate_test_name<H,W,NUM_WORKERS,_K,args...>(test::test_identifier);
        if constexpr (test::template valid<H, W, NUM_WORKERS, _K, args...>::value) {

            // initialize
            kittens::bf16 *d_re_i, *d_im_i;
            kittens::bf16 *d_re_o, *d_im_o;
            std::vector<float> re_i_ref(H*W*256);
            std::vector<float> im_i_ref(H*W*256);
            std::vector<float> re_o_ref(H*W*256);
            std::vector<float> im_o_ref(H*W*256);
            initialize(&d_re_i, &d_re_o, re_i_ref, re_o_ref);
            initialize(&d_im_i, &d_im_o, im_i_ref, im_o_ref);

            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                global_cmplx_wrapper_2d<test), kittens::bf16, H, W, NUM_WORKERS, _K, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );

            // Can't use global_wrapper_2d b/c it only accepts 2 params and we need 4 for complex-valued function
            global_cmplx_wrapper_2d<test, kittens::bf16, H, W, NUM_WORKERS, _K, args...><<<1, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(
                d_re_i, d_im_i, 
                d_re_o, d_im_o
            );

            // fill in correct results on cpu
            test::template host_func<H, W, NUM_WORKERS, _K, args...>(
                re_i_ref, im_i_ref, 
                re_o_ref, im_o_ref
            );

            // check and cleanup
            test_result re_result = validate(
                d_re_i, d_re_o, re_i_ref, re_o_ref, this_result.label + "_real", W*16, 0.02
            ); 
            test_result im_result = validate(
                d_im_i, d_im_o, im_i_ref, im_o_ref, this_result.label + "_imag", W*16, 0.02
            );

            if (re_result == test_result::PASSED && im_result == test_result::PASSED) {
                this_result.result = test_result::PASSED;
            } else {
                this_result.result = test_result::FAILED;
            }
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};

template<typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args> using cmplx_mul_sweep_size = loop_h<cmplx_mul_wrapper_2d, test, MAX_H, MAX_W, NUM_WORKERS, MAX_H, args...>;
template<typename test, int MAX_H=8, int MAX_W=8, typename... args> using cmplx_mul_sweep_size_warp = cmplx_mul_sweep_size<test, MAX_H, MAX_W, 1, args...>;

void warp::reg::tile::mul::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/register/tile/mul tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  : 
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  : 
                         INTENSITY_4 ? 16 : -1;
    cmplx_mul_sweep_size_warp<test_cmplx_mul, SIZE, SIZE, std::integral_constant<int, 1>>::run(results);
    cmplx_mul_sweep_size_warp<test_cmplx_mul, SIZE, SIZE, std::integral_constant<int, 2>>::run(results);

}

#endif